#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        printf("Error: %s:%d, ", __FILE__, __LINE__);\
        printf("code:%d, reason: %s", error, hipGetErrorString(error));\
        exit(-10 * error);\
    }\
}\


void initHostMatrix(int *h_A, int nxy)
{
    for (int i = 0; i < nxy; ++i)
    {
        h_A[i] = i;
    }
}

void printMatrix(int *h_A, int nx, int ny)
{
    for (int i = 0; i < ny; ++i) {
        for (int j = 0; j < nx; ++j) {
            printf("%d\t", h_A[i * nx + j]);
        }
        printf("\n");
    }
}

__global__ void printThreadIndex(int *d_A, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    
    printf("thread_id (%d, %d), block_id(%d, %d), coordinate(%d, %d), "
           "global index %d value %d\n", threadIdx.x, threadIdx.y,
           blockIdx.x, blockIdx.y, ix, iy, idx, d_A[idx]);
}


int main(void)
{
    //get device info
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("using device %d : %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    //set matrix dimension
    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(int);

    //malloc host memory
    int *h_A;
    h_A = (int*)malloc(nBytes);
    
    //init host matrix
    initHostMatrix(h_A, nxy);
    printMatrix(h_A, nx, ny);

    int *d_A;
    hipMalloc((void **)&d_A, nBytes);

    //transfer data to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);

    //set up execution configuration
    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    //invoke the kernel
    printThreadIndex <<<grid, block>>>(d_A, nx, ny);
    hipDeviceSynchronize();

    //free host and device memory
    hipFree(d_A);
    free(h_A);

    //reset device
    hipDeviceReset();

    return 0;
}
